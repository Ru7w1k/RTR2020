
#include <hip/hip_runtime.h>

/* helper functions for float3 */
__host__ __device__ float3 operator+(const float3 &a, const float3 &b) 
{
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__host__ __device__ float3 operator-(const float3 &a, const float3 &b) 
{
	return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__host__ __device__ float3 operator*(const float3 &a, float b) 
{
	return make_float3(a.x*b, a.y*b, a.z*b);
}

__host__ __device__ float3 operator*(float b, const float3 &a) 
{
	return make_float3(a.x*b, a.y*b, a.z*b);
}

__host__ __device__ float3 operator/(const float3 &a, float b) 
{
	return make_float3(a.x/b, a.y/b, a.z/b);
}

__host__ __device__ float3 operator/(float b, const float3 &a) 
{
	return make_float3(a.x/b, a.y/b, a.z/b);
}

__host__ __device__ float length(const float3 &a) 
{
	return sqrt(a.x*a.x + a.y*a.y + a.z*a.z);
}

__host__ __device__ float3 normalize(const float3 &a) 
{
	return a/length(a);
}

__host__ __device__ float3 cross(const float3 &a, const float3 &b)
{
	return make_float3(
		(a.y*b.z - a.z*b.y),
		(-(a.x*b.z - a.z*b.x)),
		(a.x*b.y - a.y*b.x)
	);
}

__host__ __device__ float3 make_float3(const float4 &b)
{
	return make_float3(b.x, b.y, b.z);
}


// cloth update
__global__ void cloth_kernel(float4 *pos1, float4 *pos2, float4 *vel1, float4 *vel2, unsigned int width, unsigned int height, float3 wind, float xOffset)
{
	unsigned int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	unsigned int idx = (y*width) + x;

	if (idx >= width*height) return;

	const float m = 1.0f;
	const float t = 0.000005 * 4;
	const float k = 6000.0;
	const float c = 0.95;
	const float rest_length = 1.00;
	const float rest_length_diag = 1.41;

	float3 p = make_float3(pos1[idx].x, pos1[idx].y, pos1[idx].z);
	float3 u = make_float3(vel1[idx].x, vel1[idx].y, vel1[idx].z);
	float3 F = make_float3(0.0f, -10.0f, 0.0f) * m - c * u;
	int i = 0;

	F = F + wind;

	if (true) // (vel1[idx].w >= 0.0f)
	{
		// calculate 8 connections
		// up
		if (y < height-1)
		{
			i = idx+width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// down
		if (y > 0)
		{
			i = idx-width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// left
		if (x > 0)
		{
			i = idx-1;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// right
		if (x < width-1)
		{
			i = idx+1;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}

		// lower left
		if (x > 0 && y > 0)
		{
			i = idx-1-width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// upper right
		if (x < (width-1) && y < (height-1))
		{
			i = idx+1+width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// lower right
		if (x < (width-1) && y > 0)
		{
			i = idx+1-width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// upper left
		if (x > 0 && y < (height-1))
		{
			i = idx-1+width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}

	}
	else
	{
		F = make_float3(0.0f, 0.0f, 0.0f);
	}

	// self collision!
	//int nbrs[] = {idx+width,idx-width,idx-1,idx+1,idx-1-width,idx+1+width,idx+1-width,idx-1+width};
	

	float3 a = F/m;
	float3 s = u * t + 0.5f * a * t * t;
	float3 v = u + a * t;

	
	// else if (vec3(p+s).y <= -4.0 && abs(vec3(p+s).x) < 5.5 && abs(vec3(p+s).z) < 5.5)
	// {	
	// 	s = vec3(0.0);
	// 	v = vec3(0.0);
	// }	
	

	// float force = length(F);
	// for(int i = 0; i < width*height && i!=idx; i++)
	// {
	// 	float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
	// 	float3 d = q - pos;
	// 	if(length(d) < 0.4)
	// 		v = v-force*normalize(d);
		
	// }

	// if (pos.y <= -2.0 && abs(pos.x) < 10.5 && abs(pos.z) < 10.5)
	// {	
	// 	pos = p;
	// 	v = make_float3(0.0f, 0.0f, 0.0f);
	// }
	// else 


	
	float3 op = p-make_float3(-15.0f,-4.0f,-15.0f);
	float lop = length(op);
	if (lop < 8.0)
	{	
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p-make_float3(15.0f,-4.0f,-15.0f);
	lop = length(op);
	if (lop < 8.0)
	{	
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p-make_float3(-15.0f,-4.0f,15.0f);
	lop = length(op);
	if (lop < 8.0)
	{	
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p-make_float3(15.0f,-4.0f,15.0f);
	lop = length(op);
	if (lop < 8.0)
	{	
		s.y = 0.0f;
		v.y = 0.0f;
	}

	
	// if (p.y <= -4.0 && abs(p.x) < 15.5 && abs(p.z) < 15.5)
	// {
	// 	s.y = 0.0f;
	// 	v.y = 0.0f;
	// }

	

	float3 pos = p + s;

	pos2[idx] = make_float4(pos.x, pos.y, pos.z, 1.0f);
	vel2[idx] = make_float4(v.x, v.y, v.z, vel1[idx].w);

	return;
}

__global__ void cloth_normals(float4 *pos, float3 *norm, unsigned int width, unsigned int height)
{
	unsigned int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	unsigned int idx = (y*width) + x;

	if (idx >= width*height) return;

	float3 p = make_float3(pos[idx].x, pos[idx].y, pos[idx].z);
	float3 n = make_float3(0.0f, 0.0f, 0.0f);
	float3 a, b, c;

	if (y < height-1)
	{
		c = make_float3(pos[idx+width]) - p;
		if (x < width-1)
		{
			a = make_float3(pos[idx+1]) - p;
			b = make_float3(pos[idx+width+1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
		if (x > 0)
		{
			a = c;
			b = make_float3(pos[idx+width-1]) - p;
			c = make_float3(pos[idx-1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
	}

	if (y > 0)
	{
		c = make_float3(pos[idx-width]) - p;
		if (x > 0)
		{
			a = make_float3(pos[idx-1]) - p;
			b = make_float3(pos[idx-width-1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
		if (x < width-1)
		{
			a = c;
			b = make_float3(pos[idx-width+1]) - p;
			c = make_float3(pos[idx+1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
	}

	norm[idx] = n;
}

void launchCUDAKernel(float4 *pos1, float4 *pos2, float4 *vel1, float4 *vel2, unsigned int meshWidth, unsigned int meshHeight, float3 *norm, float3 wind, float xOffset)
{
	dim3 block(16, 16, 1);
	dim3 grid(meshWidth / block.x, meshHeight / block.y, 1);

	for(int i = 0; i < 500; i++)
	{
		cloth_kernel<<<grid, block>>>(pos1, pos2, vel1, vel2, meshWidth, meshHeight, wind, xOffset);
		//cudaDeviceSynchronize();
		cloth_kernel<<<grid, block>>>(pos2, pos1, vel2, vel1, meshWidth, meshHeight, wind, xOffset);
		//cudaDeviceSynchronize();

	}
	//cudaDeviceSynchronize();
	cloth_normals<<<grid, block>>>(pos1, norm, meshWidth, meshHeight);
}

