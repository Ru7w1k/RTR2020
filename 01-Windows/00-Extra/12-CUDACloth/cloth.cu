
#include <hip/hip_runtime.h>

__global__ void sinewave_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float animTime)
{
	unsigned int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;
	u = (u * 2.0f) - 1.0f;
	v = (v * 2.0f) - 1.0f;

	float freq = 4.0f;
	float w = sinf(freq*u + animTime) * cosf(freq*v + animTime) * 0.5f;

	pos[y*width + x] = make_float4(u, w, v, 1.0f);
	return;
}

void launchCUDAKernel(float4 *pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	dim3 block(8, 1, 1);
	dim3 grid(meshWidth / block.x, meshHeight / block.y, 1);
	sinewave_vbo_kernel<<<grid, block>>>(pos, meshWidth, meshHeight, time);
}

