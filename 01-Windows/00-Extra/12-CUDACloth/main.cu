#include "hip/hip_runtime.h"
// Headers
#include <Windows.h>
#include <stdio.h>

#include <GL/glew.h>
#include <gl/GL.h>

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include "vmath.h"
#include "resource.h"

// Linker Options
#pragma comment(lib, "glew32.lib")
#pragma comment(lib, "opengl32.lib")
#pragma comment(lib, "cudart.lib")

// Defines
#define WIN_WIDTH  800
#define WIN_HEIGHT 600

#define PRIMITIVE_RESTART 0xffffff

using namespace vmath;

enum {
	AMC_ATTRIBUTE_POSITION = 0,
	AMC_ATTRIBUTE_COLOR,
	AMC_ATTRIBUTE_NORMAL,
	AMC_ATTRIBUTE_TEXCOORD0
};

// Global Variables
const int gMeshWidth = 6 * 8;
const int gMeshHeight = 6 * 8;
const int gMeshTotal = gMeshWidth * gMeshHeight;

#define MY_ARRAY_SIZE gMeshWidth*gMeshHeight*4

float4 pos[gMeshTotal] = { 0 };
float4 pos1[gMeshTotal] = { 0 };
float4 vel[gMeshTotal] = { 0 };
float4 vel1[gMeshTotal] = { 0 };

FILE  *gpFile = NULL;
bool  gbActiveWindow = false;
bool  gbIsFullScreen = false;
HDC   ghDC = NULL;
HGLRC ghRC = NULL;
HWND  ghWnd = NULL;
DWORD dwStyle;

WINDOWPLACEMENT wpPrev = { sizeof(WINDOWPLACEMENT) };

GLuint gShaderProgramObject;
struct hipGraphicsResource *graphicsResource[5] = { 0 };
GLuint vao;
GLuint vbo;
GLuint vbo_norm;
GLuint vbo_gpu[6];
GLuint vbo_index;
GLuint texCloths[2];
float animationTime = 0.0f;
bool bOnGPU = true;
bool bWind = false;
hipError_t error;
bool bAnimation = true;
GLuint mvpUniform;
mat4 perspectiveProjectionMatrix;


/* helper functions for float3 */
__host__ __device__ float3 operator+(const float3 &a, const float3 &b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ float3 operator-(const float3 &a, const float3 &b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ float3 operator*(const float3 &a, float b)
{
	return make_float3(a.x*b, a.y*b, a.z*b);
}

__host__ __device__ float3 operator*(float b, const float3 &a)
{
	return make_float3(a.x*b, a.y*b, a.z*b);
}

__host__ __device__ float3 operator/(const float3 &a, float b)
{
	return make_float3(a.x / b, a.y / b, a.z / b);
}

__host__ __device__ float3 operator/(float b, const float3 &a)
{
	return make_float3(a.x / b, a.y / b, a.z / b);
}

__host__ __device__ float length(const float3 &a)
{
	return sqrt(a.x*a.x + a.y*a.y + a.z*a.z);
}

__host__ __device__ float3 normalize(const float3 &a)
{
	return a / length(a);
}

__host__ __device__ float3 cross(const float3 &a, const float3 &b)
{
	return make_float3(
		(a.y*b.z - a.z*b.y),
		(-(a.x*b.z - a.z*b.x)),
		(a.x*b.y - a.y*b.x)
	);
}

__host__ __device__ float3 make_float3(const float4 &b)
{
	return make_float3(b.x, b.y, b.z);
}



// Global function declaration
LRESULT CALLBACK WndProc(HWND, UINT, WPARAM, LPARAM);

// WinMain
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpszCmdLine, int iCmdShow)
{
	// function declarations
	int initialize(void);
	void display(void);

	// variables 
	bool bDone = false;
	int iRet = 0;
	WNDCLASSEX wndclass;
	HWND hwnd;
	MSG msg;
	TCHAR szClassName[] = TEXT("MyApp");

	// code
	// create file for logging
	if (fopen_s(&gpFile, "log.txt", "w") != 0)
	{
		MessageBox(NULL, TEXT("Cannot Create log file!"), TEXT("Error"), MB_OK | MB_ICONERROR);
		exit(0);
	}
	else
	{
		fprintf(gpFile, "Log.txt file created...\n");
	}

	// initialization of WNDCLASSEX
	wndclass.cbSize = sizeof(WNDCLASSEX);
	wndclass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
	wndclass.cbClsExtra = 0;
	wndclass.cbWndExtra = 0;
	wndclass.lpfnWndProc = WndProc;
	wndclass.hInstance = hInstance;
	wndclass.hIcon = LoadIcon(NULL, IDI_APPLICATION);
	wndclass.hCursor = LoadCursor(NULL, IDC_ARROW);
	wndclass.hbrBackground = (HBRUSH)GetStockObject(BLACK_BRUSH);
	wndclass.lpszClassName = szClassName;
	wndclass.lpszMenuName = NULL;
	wndclass.hIconSm = LoadIcon(NULL, IDI_APPLICATION);

	// register class
	RegisterClassEx(&wndclass);

	// create window
	hwnd = CreateWindowEx(WS_EX_APPWINDOW,
		szClassName,
		TEXT("CUDA OpenGL Interoperability"),
		WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_CLIPSIBLINGS | WS_VISIBLE,
		100,
		100,
		WIN_WIDTH,
		WIN_HEIGHT,
		NULL,
		NULL,
		hInstance,
		NULL);

	ghWnd = hwnd;

	iRet = initialize();
	if (iRet == -1)
	{
		fprintf(gpFile, "ChoosePixelFormat failed...\n");
		DestroyWindow(hwnd);
	}
	else if (iRet == -2)
	{
		fprintf(gpFile, "SetPixelFormat failed...\n");
		DestroyWindow(hwnd);
	}
	else if (iRet == -3)
	{
		fprintf(gpFile, "wglCreateContext failed...\n");
		DestroyWindow(hwnd);
	}
	else if (iRet == -4)
	{
		fprintf(gpFile, "wglMakeCurrent failed...\n");
		DestroyWindow(hwnd);
	}
	else
	{
		fprintf(gpFile, "initialize() successful...\n");
	}

	ShowWindow(hwnd, iCmdShow);
	SetForegroundWindow(hwnd);
	SetFocus(hwnd);

	// Game Loop 
	while (bDone == false)
	{
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
			{
				bDone = true;
			}
			else
			{
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}
		}
		else
		{
			if (gbActiveWindow == true && bAnimation)
			{
			}
			display();
		}
	}

	return((int)msg.wParam);
}

LRESULT CALLBACK WndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam)
{
	// function declarations
	void resize(int, int);
	void uninitialize();

	void ToggleFullScreen(void);

	// code
	switch (iMsg)
	{

	case WM_SETFOCUS:
		gbActiveWindow = true;
		break;

	case WM_KILLFOCUS:
		gbActiveWindow = false;
		break;

	case WM_SIZE:
		resize(LOWORD(lParam), HIWORD(lParam));
		break;

	case WM_CLOSE:
		DestroyWindow(hwnd);
		break;

	case WM_CHAR:
		switch (wParam)
		{
		case 'G':
		case 'g':
			if (!bOnGPU)
			{
				// copy data from vertex buffer of CPU to vertex buffer of GPU
				glBindBuffer(GL_COPY_READ_BUFFER, vbo);
				glBindBuffer(GL_COPY_WRITE_BUFFER, vbo_gpu[0]);

				glCopyBufferSubData(GL_COPY_READ_BUFFER, GL_COPY_WRITE_BUFFER, 0, 0, gMeshTotal * sizeof(float));

				glBindBuffer(GL_COPY_READ_BUFFER, 0);
				glBindBuffer(GL_COPY_WRITE_BUFFER, 0);
				bOnGPU = true;
			}
			break;

		case 'W':
		case 'w':
			bWind = !bWind;
			break;

		case 'C':
		case 'c':
			if (bOnGPU)
			{
				glBindVertexArray(vao);
				glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[0]);

				vec4* p = (vec4*)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);
				memcpy_s(pos, gMeshTotal * sizeof(float4), p, gMeshTotal * sizeof(float4));
				glUnmapBuffer(GL_ARRAY_BUFFER);
				glBindBuffer(GL_ARRAY_BUFFER, 0);
				glBindVertexArray(0);
				bOnGPU = false;
			}
			break;

		case 'A':
		case 'a':
			if (bAnimation == TRUE) {
				bAnimation = FALSE;
			}
			else {
				bAnimation = TRUE;
			}
			break;
		}

	case WM_KEYDOWN:
		switch (wParam)
		{
		case VK_ESCAPE:
			DestroyWindow(hwnd);
			break;

		case 0x46:
			ToggleFullScreen();
			break;
		}
		break;

		// returned from here, to block DefWindowProc
		// We have our own painter
	case WM_ERASEBKGND:
		return(0);
		break;

	case WM_DESTROY:
		uninitialize();
		PostQuitMessage(0);
		break;
	}

	return(DefWindowProc(hwnd, iMsg, wParam, lParam));
}

void ToggleFullScreen()
{
	MONITORINFO MI;

	if (gbIsFullScreen == false)
	{
		dwStyle = GetWindowLong(ghWnd, GWL_STYLE);
		if (dwStyle & WS_OVERLAPPEDWINDOW)
		{
			MI = { sizeof(MONITORINFO) };
			if (GetWindowPlacement(ghWnd, &wpPrev)
				&& GetMonitorInfo(MonitorFromWindow(ghWnd, MONITORINFOF_PRIMARY), &MI))
			{
				SetWindowLong(ghWnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);
				SetWindowPos(ghWnd,
					HWND_TOP,
					MI.rcMonitor.left,
					MI.rcMonitor.top,
					MI.rcMonitor.right - MI.rcMonitor.left,
					MI.rcMonitor.bottom - MI.rcMonitor.top,
					SWP_NOZORDER | SWP_FRAMECHANGED);
			}
		}
		ShowCursor(FALSE);
		gbIsFullScreen = true;
	}
	else
	{
		SetWindowLong(ghWnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(ghWnd, &wpPrev);
		SetWindowPos(ghWnd,
			HWND_TOP,
			0,
			0,
			0,
			0,
			SWP_NOZORDER | SWP_FRAMECHANGED | SWP_NOMOVE | SWP_NOSIZE | SWP_NOOWNERZORDER);

		ShowCursor(TRUE);
		gbIsFullScreen = false;
	}
}

int initialize(void)
{
	// function declarations
	void resize(int, int);
	void uninitialize(void);
	BOOL loadTexture(GLuint*, TCHAR[]);


	// variable declarations
	PIXELFORMATDESCRIPTOR pfd;
	int iPixelFormatIndex;
	GLenum result;

	GLuint vertexShaderObject;
	GLuint fragmentShaderObject;

	// code
	// initialize pdf structure
	ZeroMemory((void *)&pfd, sizeof(PIXELFORMATDESCRIPTOR));
	pfd.nSize = sizeof(PIXELFORMATDESCRIPTOR);
	pfd.nVersion = 1;
	pfd.dwFlags = PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL | PFD_DOUBLEBUFFER;
	pfd.iPixelType = PFD_TYPE_RGBA;
	pfd.cColorBits = 32;
	pfd.cRedBits = 8;
	pfd.cGreenBits = 8;
	pfd.cBlueBits = 8;
	pfd.cAlphaBits = 8;
	pfd.cDepthBits = 32;

	ghDC = GetDC(ghWnd);

	iPixelFormatIndex = ChoosePixelFormat(ghDC, &pfd);
	// iPixelFormatIndex is 1 based, so 0 indicates error
	if (iPixelFormatIndex == 0)
	{
		return(-1);
	}

	if (SetPixelFormat(ghDC, iPixelFormatIndex, &pfd) == FALSE)
	{
		return(-2);
	}

	ghRC = wglCreateContext(ghDC);
	if (ghRC == NULL)
	{
		return(-3);
	}

	if (wglMakeCurrent(ghDC, ghRC) == FALSE)
	{
		return(-4);
	}

	//// C U D A /////////////////////////////////////////////////////////

	// cuda initialization
	int devCount;
	error = hipGetDeviceCount(&devCount);
	if (error != hipSuccess)
	{
		fprintf(gpFile, "hipGetDeviceCount failed..\n");
		uninitialize();
		DestroyWindow(ghWnd);
	}
	else if (devCount == 0)
	{
		fprintf(gpFile, "No CUDA device detected..\n");
		uninitialize();
		DestroyWindow(ghWnd);
	}
	else
	{
		error = hipSetDevice(0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipSetDevice failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}
	}

	//// Programable Pipeline ////////////////////////////////////////////

	result = glewInit();
	if (result != GLEW_OK) {
		fprintf(gpFile, "GLEW initialization failed..\n");
		uninitialize();
		DestroyWindow(ghWnd);
	}

	// create vertex shader object
	vertexShaderObject = glCreateShader(GL_VERTEX_SHADER);

	// vertex shader source code 
	const GLchar *vertexShaderSourceCode = (GLchar *)
		"#version 450 core" \
		"\n" \

		"in vec4 position;" \
		"in vec3 normal;" \
		"in vec2 texcoord;" \

		"uniform float front = 1.0f;" \
		"uniform mat4 u_m_matrix;" \
		"uniform mat4 u_v_matrix;" \
		"uniform mat4 u_p_matrix;" \
		"uniform vec4 u_light_position = vec4(0.0f, 5.0f, 0.0f, 1.0f);" \

		"out vec3 tnorm;" \
		"out vec3 light_direction;" \
		"out vec3 viewer_vector;" \
		"out vec2 out_Texcoord;" \

		"void main()" \
		"{" \

		"   vec4 eye_coordinates = u_v_matrix * u_m_matrix * position;" \
		"   tnorm = mat3(u_v_matrix * u_m_matrix) * normal * front;" \
		"   light_direction = vec3(u_light_position - eye_coordinates);" \
		"   float tn_dot_ldir = max(dot(tnorm, light_direction), 0.0);" \
		"   viewer_vector = vec3(-eye_coordinates.xyz);" \

		"	gl_Position = u_p_matrix * u_v_matrix * u_m_matrix * vec4(position.xyz , 1.0);" \
		"   out_Texcoord = texcoord;" \
		"}";

	// attach source code to vertex shader
	glShaderSource(vertexShaderObject, 1, (const GLchar **)&vertexShaderSourceCode, NULL);

	// compile vertex shader source code
	glCompileShader(vertexShaderObject);

	// compilation errors 
	GLint iShaderCompileStatus = 0;
	GLint iInfoLogLength = 0;
	GLchar *szInfoLog = NULL;

	glGetShaderiv(vertexShaderObject, GL_COMPILE_STATUS, &iShaderCompileStatus);
	if (iShaderCompileStatus == GL_FALSE)
	{
		glGetShaderiv(vertexShaderObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength > 0)
		{
			szInfoLog = (GLchar *)malloc(iInfoLogLength);
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(vertexShaderObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);

				fprintf(gpFile, "Vertex Shader Compiler Info Log: %s", szInfoLog);
				free(szInfoLog);
				uninitialize();
				DestroyWindow(ghWnd);
			}
		}
	}

	// create fragment shader object
	fragmentShaderObject = glCreateShader(GL_FRAGMENT_SHADER);

	// fragment shader source code
	const GLchar *fragmentShaderSourceCode = (GLchar *)
		"#version 450 core" \
		"\n" \

		"in vec3 tnorm;" \
		"in vec3 light_direction;" \
		"in vec3 viewer_vector;" \
		"in vec2 out_Texcoord;" \

		"uniform vec3 u_la = vec3(0.4, 0.4, 0.4);" \
		"uniform vec3 u_ld = vec3(0.8, 0.8, 0.8);" \
		"uniform vec3 u_ls = vec3(1.0, 1.0, 1.0);" \
		"uniform vec3 u_ka = vec3(0.4, 0.4, 0.4);" \
		"uniform vec3 u_kd = vec3(0.8, 0.8, 0.8);" \
		"uniform vec3 u_ks = vec3(1.0, 1.0, 1.0);" \
		"uniform float u_shininess = 25.0;" \

		"uniform sampler2D u_sampler;" \

		"out vec4 FragColor;" \

		"void main (void)" \
		"{" \
		"   vec3 ntnorm = normalize(tnorm);" \
		"   vec3 nlight_direction = normalize(light_direction);" \
		"   vec3 nviewer_vector = normalize(viewer_vector);" \
		"   vec3 reflection_vector = reflect(-nlight_direction, ntnorm);" \
		"   float tn_dot_ldir = max(dot(ntnorm, nlight_direction), 0.0);" \

		"   vec3 ambient  = u_la * u_ka;" \
		"   vec3 diffuse  = u_ld * u_kd * tn_dot_ldir;" \
		"   vec3 specular = u_ls * u_ks * pow(max(dot(reflection_vector, nviewer_vector), 0.0), u_shininess);" \

		"   vec3 phong_ads_light = ambient + diffuse;" \

		"   FragColor = vec4(phong_ads_light, 1.0) * texture(u_sampler, out_Texcoord);" \
		"}";

	// attach source code to fragment shader
	glShaderSource(fragmentShaderObject, 1, (const GLchar **)&fragmentShaderSourceCode, NULL);

	// compile fragment shader source code
	glCompileShader(fragmentShaderObject);

	// compile errors
	iShaderCompileStatus = 0;
	iInfoLogLength = 0;
	szInfoLog = NULL;

	glGetShaderiv(fragmentShaderObject, GL_COMPILE_STATUS, &iShaderCompileStatus);
	if (iShaderCompileStatus == GL_FALSE)
	{
		glGetShaderiv(fragmentShaderObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength > 0)
		{
			szInfoLog = (GLchar *)malloc(iInfoLogLength);
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(fragmentShaderObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);

				fprintf(gpFile, "Fragment Shader Compiler Info Log: %s", szInfoLog);
				free(szInfoLog);
				uninitialize();
				DestroyWindow(ghWnd);
			}
		}
	}

	// create shader program object 
	gShaderProgramObject = glCreateProgram();

	// attach vertex shader to shader program
	glAttachShader(gShaderProgramObject, vertexShaderObject);

	// attach fragment shader to shader program
	glAttachShader(gShaderProgramObject, fragmentShaderObject);

	// pre-linking binding to vertex attribute
	glBindAttribLocation(gShaderProgramObject, AMC_ATTRIBUTE_POSITION, "position");
	glBindAttribLocation(gShaderProgramObject, AMC_ATTRIBUTE_NORMAL, "normal");
	glBindAttribLocation(gShaderProgramObject, AMC_ATTRIBUTE_TEXCOORD0, "texcoord");

	// link the shader program
	glLinkProgram(gShaderProgramObject);

	// linking errors
	GLint iProgramLinkStatus = 0;
	iInfoLogLength = 0;
	szInfoLog = NULL;

	glGetProgramiv(gShaderProgramObject, GL_LINK_STATUS, &iProgramLinkStatus);
	if (iProgramLinkStatus == GL_FALSE)
	{
		glGetProgramiv(gShaderProgramObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength > 0)
		{
			szInfoLog = (GLchar *)malloc(iInfoLogLength);
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetProgramInfoLog(gShaderProgramObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);

				fprintf(gpFile, ("Shader Program Linking Info Log: %s"), szInfoLog);
				free(szInfoLog);
				uninitialize();
				DestroyWindow(ghWnd);
			}
		}
	}

	// post-linking retrieving uniform locations

	///// cloth mesh coordinates generation ///////////////////////////////////////
	int i, j;

	vec4 *initial_positions = new vec4[gMeshTotal];
	vec4 *initial_velocities = new vec4[gMeshTotal];
	vec3 *initial_normals = new vec3[gMeshTotal];
	vec2 *initial_texcoords = new vec2[gMeshTotal];

	int n = 0;

	for (j = 0; j < gMeshHeight; j++)
	{
		float fj = (float)j / (float)gMeshHeight;
		for (i = 0; i < gMeshWidth; i++)
		{
			float fi = (float)i / (float)gMeshWidth;

			initial_positions[n] = vec4((fi - 0.5f) * (float)gMeshWidth,
				10.0f,
				(fj - 0.5f) * (float)gMeshHeight,
				1.0);

			initial_velocities[n] = vec4(0.0f);
			initial_normals[n] = vec3(0.0f);

			// texture coords
			initial_texcoords[n][0] = fi * 5.0f;
			initial_texcoords[n][1] = fj * 5.0f;

			n++;

		}
	}

	// create vao
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

	// vertex positions
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), initial_positions, GL_DYNAMIC_DRAW);

	glGenBuffers(1, &vbo_norm);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_norm);
	glBufferData(GL_ARRAY_BUFFER, gMeshTotal * 3 * sizeof(float), initial_velocities, GL_DYNAMIC_DRAW);

	// vertex positions
	glGenBuffers(6, vbo_gpu);

	// pos1 and pos2
	for (int i = 0; i < 2; i++)
	{
		glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[i]);
		glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), initial_positions, GL_DYNAMIC_DRAW);

		// register our vbo with cuda graphics resource
		error = hipGraphicsGLRegisterBuffer(&graphicsResource[i], vbo_gpu[i], cudaGraphicsMapFlagsWriteDiscard);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsGLRegisterBuffer failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}
	}

	// vel1 and vel2
	for (int i = 2; i < 4; i++)
	{
		glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[i]);
		glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), initial_velocities, GL_DYNAMIC_DRAW);

		// register our vbo with cuda graphics resource
		error = hipGraphicsGLRegisterBuffer(&graphicsResource[i], vbo_gpu[i], cudaGraphicsMapFlagsWriteDiscard);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsGLRegisterBuffer failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}
	}

	// normals
	glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[4]);
	glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), initial_normals, GL_DYNAMIC_DRAW);

	// register our vbo with cuda graphics resource
	error = hipGraphicsGLRegisterBuffer(&graphicsResource[4], vbo_gpu[4], cudaGraphicsMapFlagsWriteDiscard);
	if (error != hipSuccess)
	{
		fprintf(gpFile, "hipGraphicsGLRegisterBuffer failed..\n");
		uninitialize();
		DestroyWindow(ghWnd);
	}

	// texcoords
	glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[5]);
	glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), initial_texcoords, GL_DYNAMIC_DRAW);
	glVertexAttribPointer(AMC_ATTRIBUTE_TEXCOORD0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(AMC_ATTRIBUTE_TEXCOORD0);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	// index buffer for cloth mesh
	int lines = (gMeshWidth * (gMeshHeight - 1)) + gMeshWidth;

	glGenBuffers(1, &vbo_index);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_index);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, lines * 2 * sizeof(int), NULL, GL_STATIC_DRAW);

	int* e = (int*)glMapBufferRange(GL_ELEMENT_ARRAY_BUFFER, 0, lines * 2 * sizeof(int), GL_MAP_WRITE_BIT | GL_MAP_INVALIDATE_BUFFER_BIT);

	// triangle mesh
	for (j = 0; j < gMeshHeight - 1; j++)
	{
		for (i = 0; i < gMeshWidth; i++)
		{
			*e++ = j * gMeshWidth + i;
			*e++ = (1 + j) * gMeshWidth + i;
		}
		*e++ = PRIMITIVE_RESTART;
	}

	glUnmapBuffer(GL_ELEMENT_ARRAY_BUFFER);

	delete[]initial_positions;
	delete[]initial_velocities;
	delete[]initial_normals;
	delete[]initial_texcoords;

	//////////////////////////////////////////////////////////////////////

	// clear the depth buffer
	glClearDepth(1.0f);

	// primitive restart
	glEnable(GL_PRIMITIVE_RESTART);
	glPrimitiveRestartIndex(PRIMITIVE_RESTART);

	// enable depth
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LEQUAL);

	// enable blend
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	// clear the screen by OpenGL
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

	// textures
	glEnable(GL_TEXTURE_2D);
	loadTexture(&texCloths[0], MAKEINTRESOURCE(IDBITMAP_CLOTH1));
	loadTexture(&texCloths[1], MAKEINTRESOURCE(IDBITMAP_CLOTH2));

	perspectiveProjectionMatrix = mat4::identity();

	// warm-up call to resize
	resize(WIN_WIDTH, WIN_HEIGHT);

	return(0);
}

void resize(int width, int height)
{
	if (height == 0)
	{
		height = 1;
	}

	glViewport(0, 0, (GLsizei)width, (GLsizei)height);

	perspectiveProjectionMatrix = perspective(45.0, (float)width / (float)height, 0.1f, 100.0f);
}


void display(void)
{
	void uninitialize(void);
	void launchCUDAKernel(float4 *, float4 *, float4 *, float4 *, unsigned int, unsigned int, float3 *, float3, float);
	void launchCPUKernel(unsigned int, unsigned int, float3);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// use shader program
	glUseProgram(gShaderProgramObject);

	mat4 mMatrix = mat4::identity();
	//mMatrix *= rotate(0.0f, 100.0f*sinf(t), 0.0f);

	mat4 vMatrix = mat4::identity();
	static float angle = 0.0f;
	angle += 0.005f;
	vMatrix *= lookat(
		vec3(60.0f*sinf(angle), 0.0f, 60.0f*cosf(angle)),
		vec3(0.0f, 0.0f, 0.0f),
		vec3(0.0f, 1.0f, 0.0f));

	glUniformMatrix4fv(glGetUniformLocation(gShaderProgramObject, "u_m_matrix"), 1, GL_FALSE, mMatrix);
	glUniformMatrix4fv(glGetUniformLocation(gShaderProgramObject, "u_v_matrix"), 1, GL_FALSE, vMatrix);
	glUniformMatrix4fv(glGetUniformLocation(gShaderProgramObject, "u_p_matrix"), 1, GL_FALSE, perspectiveProjectionMatrix);

	float3 wind = make_float3(0.0f, 0.0f, 0.0f);
	if (bWind) wind = make_float3(5.0f, 0.0f, 0.0f);


	glBindVertexArray(vao);

	if (bOnGPU)
	{
		// 1. map with the resource
		error = hipGraphicsMapResources(1, &graphicsResource[0], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "cudaGraphicsMapResource 0 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsMapResources(1, &graphicsResource[1], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "cudaGraphicsMapResource 1 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsMapResources(1, &graphicsResource[2], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "cudaGraphicsMapResource 2 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsMapResources(1, &graphicsResource[3], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "cudaGraphicsMapResource 3 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsMapResources(1, &graphicsResource[4], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "cudaGraphicsMapResource 4 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		// 2. get pointer to mapped resource
		float4 *ppos1 = NULL;
		float4 *ppos2 = NULL;
		float4 *pvel1 = NULL;
		float4 *pvel2 = NULL;
		float3 *norm = NULL;

		size_t byteCount;
		error = hipGraphicsResourceGetMappedPointer((void **)&ppos1, &byteCount, graphicsResource[0]);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer ppos1 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsResourceGetMappedPointer((void **)&ppos2, &byteCount, graphicsResource[1]);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer ppos2 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsResourceGetMappedPointer((void **)&pvel1, &byteCount, graphicsResource[2]);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer pvel1 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsResourceGetMappedPointer((void **)&pvel2, &byteCount, graphicsResource[3]);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer pvel2 failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsResourceGetMappedPointer((void **)&norm, &byteCount, graphicsResource[4]);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer norm failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}


		// 3. launch the CUDA kernel
		static float xOffset = 0.0f;
		launchCUDAKernel(ppos1, ppos2, pvel1, pvel2, gMeshWidth, gMeshHeight, norm, wind, xOffset);
		xOffset += 0.01f;

		// 4. unmap the resource
		error = hipGraphicsUnmapResources(1, &graphicsResource[0], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsUnmapResources(1, &graphicsResource[1], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsUnmapResources(1, &graphicsResource[2], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsUnmapResources(1, &graphicsResource[3], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

		error = hipGraphicsUnmapResources(1, &graphicsResource[4], 0);
		if (error != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources failed..\n");
			uninitialize();
			DestroyWindow(ghWnd);
		}

	}
	else
	{
		launchCPUKernel(gMeshWidth, gMeshHeight, wind);

		glBindBuffer(GL_ARRAY_BUFFER, vbo);
		glBufferData(GL_ARRAY_BUFFER, MY_ARRAY_SIZE * sizeof(float), pos, GL_DYNAMIC_DRAW);
	}

	// bind to the respective buffer
	if (bOnGPU) glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[0]);
	else glBindBuffer(GL_ARRAY_BUFFER, vbo);

	glVertexAttribPointer(AMC_ATTRIBUTE_POSITION, 4, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(AMC_ATTRIBUTE_POSITION);

	if (bOnGPU) glBindBuffer(GL_ARRAY_BUFFER, vbo_gpu[4]);
	else glBindBuffer(GL_ARRAY_BUFFER, vbo_norm);

	glVertexAttribPointer(AMC_ATTRIBUTE_NORMAL, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(AMC_ATTRIBUTE_NORMAL);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, texCloths[1]);

	int lines = (gMeshWidth * (gMeshHeight - 1)) + gMeshWidth;
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_index);

	// draw now!
	// front side
	glUniform1f(glGetUniformLocation(gShaderProgramObject, "front"), -1.0f);
	glCullFace(GL_BACK);
	glDrawElements(GL_TRIANGLE_STRIP, lines * 2, GL_UNSIGNED_INT, NULL);

	// back side
	glUniform1f(glGetUniformLocation(gShaderProgramObject, "front"), -1.0f);
	glCullFace(GL_FRONT);
	glDrawElements(GL_TRIANGLE_STRIP, lines * 2, GL_UNSIGNED_INT, NULL);

	glBindVertexArray(0);

	//////////////////////////////////////////////////////////////////////////////////////////

	// unuse program
	glUseProgram(0);

	SwapBuffers(ghDC);
	animationTime += 0.05f;
}

void uninitialize(void)
{
	if (vbo_gpu)
	{
		glDeleteBuffers(4, vbo_gpu);
		for (int i = 0; i < 4; i++)
			vbo_gpu[i] = 0;
	}

	if (vbo)
	{
		glDeleteBuffers(1, &vbo);
		vbo = 0;
	}

	if (vao)
	{
		glDeleteVertexArrays(1, &vao);
		vao = 0;
	}

	for (int i = 0; i < 5; i++)
	{
		if (graphicsResource[i])
		{
			hipGraphicsUnregisterResource(graphicsResource[i]);
			graphicsResource[i] = NULL;
		}
	}

	if (gShaderProgramObject)
	{
		GLsizei shaderCount;
		GLsizei shaderNumber;

		glUseProgram(gShaderProgramObject);
		glGetProgramiv(gShaderProgramObject, GL_ATTACHED_SHADERS, &shaderCount);

		GLuint *pShaders = (GLuint *)malloc(sizeof(GLuint) * shaderCount);
		if (pShaders)
		{
			glGetAttachedShaders(gShaderProgramObject, shaderCount, &shaderCount, pShaders);

			for (shaderNumber = 0; shaderNumber < shaderCount; shaderNumber++)
			{
				// detach shader
				glDetachShader(gShaderProgramObject, pShaders[shaderNumber]);

				// delete shader
				glDeleteShader(pShaders[shaderNumber]);
				pShaders[shaderNumber] = 0;
			}
			free(pShaders);
		}

		glDeleteProgram(gShaderProgramObject);
		gShaderProgramObject = 0;
		glUseProgram(0);

	}

	// fullscreen check
	if (gbIsFullScreen == true)
	{
		SetWindowLong(ghWnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(ghWnd, &wpPrev);
		SetWindowPos(ghWnd,
			HWND_TOP,
			0,
			0,
			0,
			0,
			SWP_NOZORDER | SWP_FRAMECHANGED | SWP_NOMOVE | SWP_NOSIZE | SWP_NOOWNERZORDER);

		ShowCursor(TRUE);
	}

	// break the current context
	if (wglGetCurrentContext() == ghRC)
	{
		wglMakeCurrent(NULL, NULL);
	}

	if (ghRC)
	{
		wglDeleteContext(ghRC);
	}

	if (ghDC)
	{
		ReleaseDC(ghWnd, ghDC);
		ghDC = NULL;
	}

	if (gpFile)
	{
		fprintf(gpFile, "Log file is closed...\n");
		fclose(gpFile);
		gpFile = NULL;
	}
}

void launchCPUKernel(unsigned int width, unsigned int height, float3 wind)
{
	vec3 make_vec3(float4);
	vec3 make_vec3(float *);

	const float m = 1.0f;
	const float t = 0.000005 * 4;
	const float k = 6000.0;
	const float c = 0.95;
	const float rest_length = 1.00;
	const float rest_length_diag = 1.41;

	// latest position in global pos

	float4 *ppos1 = pos;
	float4 *ppos2 = pos1;
	float4 *pvel1 = vel;
	float4 *pvel2 = vel1;

	for (int count = 0; count < 1000; count++)
	{
		for (unsigned int x = 0; x < width; x++)
		{
			for (unsigned int y = 0; y < height; y++)
			{
				unsigned int idx = (y*width) + x;
				float3 p = make_float3(pos1[idx].x, pos1[idx].y, pos1[idx].z);
				float3 u = make_float3(vel1[idx].x, vel1[idx].y, vel1[idx].z);
				float3 F = make_float3(0.0f, -10.0f, 0.0f) * m - c * u;
				int i = 0;

				F = F + wind;

				if (true) // (vel1[idx].w >= 0.0f)
				{
					// calculate 8 connections
					// up
					if (y < height - 1)
					{
						i = idx + width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length - x) * normalize(d);
					}
					// down
					if (y > 0)
					{
						i = idx - width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length - x) * normalize(d);
					}
					// left
					if (x > 0)
					{
						i = idx - 1;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length - x) * normalize(d);
					}
					// right
					if (x < width - 1)
					{
						i = idx + 1;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length - x) * normalize(d);
					}

					// lower left
					if (x > 0 && y > 0)
					{
						i = idx - 1 - width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length_diag - x) * normalize(d);
					}
					// upper right
					if (x < (width - 1) && y < (height - 1))
					{
						i = idx + 1 + width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length_diag - x) * normalize(d);
					}
					// lower right
					if (x < (width - 1) && y > 0)
					{
						i = idx + 1 - width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length_diag - x) * normalize(d);
					}
					// upper left
					if (x > 0 && y < (height - 1))
					{
						i = idx - 1 + width;
						float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
						float3 d = q - p;
						float x = length(d);
						F = F + -k * (rest_length_diag - x) * normalize(d);
					}

				}
				else
				{
					F = make_float3(0.0f, 0.0f, 0.0f);
				}

				// self collision!
				//int nbrs[] = {idx+width,idx-width,idx-1,idx+1,idx-1-width,idx+1+width,idx+1-width,idx-1+width};


				float3 a = F / m;
				float3 s = u * t + 0.5f * a * t * t;
				float3 v = u + a * t;


				// else if (vec3(p+s).y <= -4.0 && abs(vec3(p+s).x) < 5.5 && abs(vec3(p+s).z) < 5.5)
				// {	
				// 	s = vec3(0.0);
				// 	v = vec3(0.0);
				// }	


				// float force = length(F);
				// for(int i = 0; i < width*height && i!=idx; i++)
				// {
				// 	float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
				// 	float3 d = q - pos;
				// 	if(length(d) < 0.4)
				// 		v = v-force*normalize(d);

				// }

				// if (pos.y <= -2.0 && abs(pos.x) < 10.5 && abs(pos.z) < 10.5)
				// {	
				// 	pos = p;
				// 	v = make_float3(0.0f, 0.0f, 0.0f);
				// }
				// else 



				float3 op = p - make_float3(-15.0f, -4.0f, -15.0f);
				float lop = length(op);
				if (lop < 8.0)
				{
					s.y = 0.0f;
					v.y = 0.0f;
				}

				op = p - make_float3(15.0f, -4.0f, -15.0f);
				lop = length(op);
				if (lop < 8.0)
				{
					s.y = 0.0f;
					v.y = 0.0f;
				}

				op = p - make_float3(-15.0f, -4.0f, 15.0f);
				lop = length(op);
				if (lop < 8.0)
				{
					s.y = 0.0f;
					v.y = 0.0f;
				}

				op = p - make_float3(15.0f, -4.0f, 15.0f);
				lop = length(op);
				if (lop < 8.0)
				{
					s.y = 0.0f;
					v.y = 0.0f;
				}


				// if (p.y <= -4.0 && abs(p.x) < 15.5 && abs(p.z) < 15.5)
				// {
				// 	s.y = 0.0f;
				// 	v.y = 0.0f;
				// }



				float3 pos = p + s;

				ppos2[idx] = make_float4(pos.x, pos.y, pos.z, 1.0f);
				pvel2[idx] = make_float4(v.x, v.y, v.z, vel1[idx].w);
			}
		}
	}

	fprintf(gpFile, "\nCalculating normals!");

	// normals
	float3 *norm = new float3[gMeshTotal];
	for (int x = 0; x < width; x++)
	{
		for (int y = 0; y < height; y++)
		{
			unsigned int idx = (y*width) + x;

			float3 p = make_float3(pos[idx].x, pos[idx].y, pos[idx].z);
			float3 n = make_float3(0.0f, 0.0f, 0.0f);
			float3 a, b, c;

			if (y < height - 1)
			{
				c = make_float3(pos[idx + width]) - p;
				if (x < width - 1)
				{
					a = make_float3(pos[idx + 1]) - p;
					b = make_float3(pos[idx + width + 1]) - p;
					n = n + cross(a, b);
					n = n + cross(b, c);
				}
				if (x > 0)
				{
					a = c;
					b = make_float3(pos[idx + width - 1]) - p;
					c = make_float3(pos[idx - 1]) - p;
					n = n + cross(a, b);
					n = n + cross(b, c);
				}
			}

			if (y > 0)
			{
				c = make_float3(pos[idx - width]) - p;
				if (x > 0)
				{
					a = make_float3(pos[idx - 1]) - p;
					b = make_float3(pos[idx - width - 1]) - p;
					n = n + cross(a, b);
					n = n + cross(b, c);
				}
				if (x < width - 1)
				{
					a = c;
					b = make_float3(pos[idx - width + 1]) - p;
					c = make_float3(pos[idx + 1]) - p;
					n = n + cross(a, b);
					n = n + cross(b, c);
				}
			}

			norm[idx] = n;
		}
	}

	glBindBuffer(GL_ARRAY_BUFFER, vbo_norm);
	glBufferData(GL_ARRAY_BUFFER, gMeshTotal * 3 * sizeof(float), norm, GL_DYNAMIC_DRAW);

	delete[]norm;
}

// Convert image resource to image data
BOOL loadTexture(GLuint* texture, TCHAR imageResourceID[])
{
	// variables
	HBITMAP hBitmap = NULL;
	BITMAP bmp;
	BOOL bStatus = false;

	// data
	hBitmap = (HBITMAP)LoadImage(GetModuleHandle(NULL),
		imageResourceID,
		IMAGE_BITMAP,
		0, 0,
		LR_CREATEDIBSECTION
	);

	if (hBitmap)
	{
		bStatus = TRUE;
		GetObject(hBitmap, sizeof(BITMAP), &bmp);

		glPixelStorei(GL_UNPACK_ALIGNMENT, 4);
		glGenTextures(1, texture);
		glBindTexture(GL_TEXTURE_2D, *texture);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, bmp.bmWidth, bmp.bmHeight, 0, GL_BGR, GL_UNSIGNED_BYTE, bmp.bmBits);
		glGenerateMipmap(GL_TEXTURE_2D);

		glBindTexture(GL_TEXTURE_2D, 0);

		DeleteObject(hBitmap);
	}

	return bStatus;
}


// cloth update
__global__ void cloth_kernel(float4 *pos1, float4 *pos2, float4 *vel1, float4 *vel2, unsigned int width, unsigned int height, float3 wind, float xOffset)
{
	unsigned int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	unsigned int idx = (y*width) + x;

	if (idx >= width * height) return;

	const float m = 1.0f;
	const float t = 0.000005 * 4;
	const float k = 6000.0;
	const float c = 0.95;
	const float rest_length = 1.00;
	const float rest_length_diag = 1.41;

	float3 p = make_float3(pos1[idx].x, pos1[idx].y, pos1[idx].z);
	float3 u = make_float3(vel1[idx].x, vel1[idx].y, vel1[idx].z);
	float3 F = make_float3(0.0f, -10.0f, 0.0f) * m - c * u;
	int i = 0;

	F = F + wind;

	if (true) // (vel1[idx].w >= 0.0f)
	{
		// calculate 8 connections
		// up
		if (y < height - 1)
		{
			i = idx + width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// down
		if (y > 0)
		{
			i = idx - width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// left
		if (x > 0)
		{
			i = idx - 1;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}
		// right
		if (x < width - 1)
		{
			i = idx + 1;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length - x) * normalize(d);
		}

		// lower left
		if (x > 0 && y > 0)
		{
			i = idx - 1 - width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// upper right
		if (x < (width - 1) && y < (height - 1))
		{
			i = idx + 1 + width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// lower right
		if (x < (width - 1) && y > 0)
		{
			i = idx + 1 - width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}
		// upper left
		if (x > 0 && y < (height - 1))
		{
			i = idx - 1 + width;
			float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
			float3 d = q - p;
			float x = length(d);
			F = F + -k * (rest_length_diag - x) * normalize(d);
		}

	}
	else
	{
		F = make_float3(0.0f, 0.0f, 0.0f);
	}

	// self collision!
	//int nbrs[] = {idx+width,idx-width,idx-1,idx+1,idx-1-width,idx+1+width,idx+1-width,idx-1+width};


	float3 a = F / m;
	float3 s = u * t + 0.5f * a * t * t;
	float3 v = u + a * t;


	// else if (vec3(p+s).y <= -4.0 && abs(vec3(p+s).x) < 5.5 && abs(vec3(p+s).z) < 5.5)
	// {	
	// 	s = vec3(0.0);
	// 	v = vec3(0.0);
	// }	


	// float force = length(F);
	// for(int i = 0; i < width*height && i!=idx; i++)
	// {
	// 	float3 q = make_float3(pos1[i].x, pos1[i].y, pos1[i].z);
	// 	float3 d = q - pos;
	// 	if(length(d) < 0.4)
	// 		v = v-force*normalize(d);

	// }

	// if (pos.y <= -2.0 && abs(pos.x) < 10.5 && abs(pos.z) < 10.5)
	// {	
	// 	pos = p;
	// 	v = make_float3(0.0f, 0.0f, 0.0f);
	// }
	// else 



	float3 op = p - make_float3(-15.0f, -4.0f, -15.0f);
	float lop = length(op);
	if (lop < 8.0)
	{
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p - make_float3(15.0f, -4.0f, -15.0f);
	lop = length(op);
	if (lop < 8.0)
	{
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p - make_float3(-15.0f, -4.0f, 15.0f);
	lop = length(op);
	if (lop < 8.0)
	{
		s.y = 0.0f;
		v.y = 0.0f;
	}

	op = p - make_float3(15.0f, -4.0f, 15.0f);
	lop = length(op);
	if (lop < 8.0)
	{
		s.y = 0.0f;
		v.y = 0.0f;
	}


	// if (p.y <= -4.0 && abs(p.x) < 15.5 && abs(p.z) < 15.5)
	// {
	// 	s.y = 0.0f;
	// 	v.y = 0.0f;
	// }



	float3 pos = p + s;

	pos2[idx] = make_float4(pos.x, pos.y, pos.z, 1.0f);
	vel2[idx] = make_float4(v.x, v.y, v.z, vel1[idx].w);

	return;
}

__global__ void cloth_normals(float4 *pos, float3 *norm, unsigned int width, unsigned int height)
{
	unsigned int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	unsigned int idx = (y*width) + x;

	if (idx >= width * height) return;

	float3 p = make_float3(pos[idx].x, pos[idx].y, pos[idx].z);
	float3 n = make_float3(0.0f, 0.0f, 0.0f);
	float3 a, b, c;

	if (y < height - 1)
	{
		c = make_float3(pos[idx + width]) - p;
		if (x < width - 1)
		{
			a = make_float3(pos[idx + 1]) - p;
			b = make_float3(pos[idx + width + 1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
		if (x > 0)
		{
			a = c;
			b = make_float3(pos[idx + width - 1]) - p;
			c = make_float3(pos[idx - 1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
	}

	if (y > 0)
	{
		c = make_float3(pos[idx - width]) - p;
		if (x > 0)
		{
			a = make_float3(pos[idx - 1]) - p;
			b = make_float3(pos[idx - width - 1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
		if (x < width - 1)
		{
			a = c;
			b = make_float3(pos[idx - width + 1]) - p;
			c = make_float3(pos[idx + 1]) - p;
			n = n + cross(a, b);
			n = n + cross(b, c);
		}
	}

	norm[idx] = n;
}

void launchCUDAKernel(float4 *pos1, float4 *pos2, float4 *vel1, float4 *vel2, unsigned int meshWidth, unsigned int meshHeight, float3 *norm, float3 wind, float xOffset)
{
	dim3 block(16, 16, 1);
	dim3 grid(meshWidth / block.x, meshHeight / block.y, 1);

	for (int i = 0; i < 500; i++)
	{
		cloth_kernel << <grid, block >> > (pos1, pos2, vel1, vel2, meshWidth, meshHeight, wind, xOffset);
		//hipDeviceSynchronize();
		cloth_kernel << <grid, block >> > (pos2, pos1, vel2, vel1, meshWidth, meshHeight, wind, xOffset);
		//hipDeviceSynchronize();

	}
	//hipDeviceSynchronize();
	cloth_normals << <grid, block >> > (pos1, norm, meshWidth, meshHeight);
}


