#include "hip/hip_runtime.h"
// Headers

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>

#include "helper_timer.h"

// Global Variables
float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;
float *gold = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

float timeOnCPU;
float timeOnGPU;

// Kernel
__global__ void VecAdd(float *in1, float *in2, float *out, int len)
{
	// calculate the current thread index 
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// calculate, if the thread is within the range of input
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}

}

// main function
int main(int argc, char **argv)
{
	// function declarations
	void vecAddCPU(float *, float *, float *, int);
	void randomInit(float *, int);
	void cleanup(void);

	// variables
	int iArraySize = 11444777;

	int size = sizeof(float) * iArraySize;

	// allocate memory on host
	hostInput1 = (float *)malloc(size);
	if (!hostInput1)
	{
		printf("Out Of Memory on Host!\nTerminating...\n\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float *)malloc(size);
	if (!hostInput2)
	{
		printf("Out Of Memory on Host!\nTerminating...\n\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float *)malloc(size);
	if (!hostOutput)
	{
		printf("Out Of Memory on Host!\nTerminating...\n\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float *)malloc(size);
	if (!gold)
	{
		printf("Out Of Memory on Host!\nTerminating...\n\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// allcate memory on device
	hipError_t cuda_error = hipSuccess;
	cuda_error = hipMalloc((void**)&deviceInput1, size);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Allocate Memory on Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	cuda_error = hipMalloc((void**)&deviceInput2, size);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Allocate Memory on Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	cuda_error = hipMalloc((void**)&deviceOutput, size);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Allocate Memory on Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}


	// Initialize the input arrays!
	randomInit(hostInput1, iArraySize);
	randomInit(hostInput2, iArraySize);

	// timer
	StopWatchInterface *stopwatch = NULL;
	sdkCreateTimer(&stopwatch);

	// run on HOST!
	sdkStartTimer(&stopwatch);
	vecAddCPU(hostInput1, hostInput2, gold, iArraySize);
	sdkStopTimer(&stopwatch);
	timeOnCPU = sdkGetTimerValue(&stopwatch);
	sdkDeleteTimer(&stopwatch);


	// print the result!
	printf("Time on CPU: %f ms\n", timeOnCPU);
	
	
	// Copy data to Device!
	cuda_error = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Copy Memory From Host to Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	cuda_error = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Copy Memory From Host to Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}


	// Kernel Configuration
	dim3 GridDim = dim3(ceil(iArraySize / 256.0), 1, 1);
	dim3 BlockDim = dim3(256, 1, 1);

	// time the execution
	stopwatch = NULL;
	sdkCreateTimer(&stopwatch);

	// Let's run!
	sdkStartTimer(&stopwatch);
	VecAdd <<<GridDim, BlockDim >>> (deviceInput1, deviceInput2, deviceOutput, iArraySize);
	sdkStopTimer(&stopwatch);

	// Copy Result from Device Memory to Host Memory
	cuda_error = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if (cuda_error != hipSuccess)
	{
		printf("Cannot Copy Memory From Host to Device!\nError: %s\nFile Name : %s, Line No: %d\n\n", hipGetErrorString(cuda_error), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	timeOnGPU = sdkGetTimerValue(&stopwatch);
	sdkDeleteTimer(&stopwatch);
	
	// print the result!
	printf("Time on GPU: %f ms\n", timeOnGPU);

	float epsilon = 0.000001f;
	bool bAccurate = true;
	int index = 0;
	
	for (int i = 0; i < iArraySize; i++)
	{
		if (fabs(gold[i] - hostOutput[i]) > epsilon)
		{
			bAccurate = false;
			index = i;
			break;
		}
	}

	if (bAccurate)
		printf("The Results are accurate upto %f %%\n\n", epsilon);
	else
		printf("The Results are not accurate, breaking index was: %d\n\n", index);

	return(0);
}


void randomInit(float *arr, int size)
{
	float fScale = 1.0 / (float)RAND_MAX;

	for (int i = 0; i < size; i++)
	{
		arr[i] = fScale * rand();
	}
}
void vecAddCPU(float *in1, float *in2, float *out, int size)
{
	for (int i = 0; i < size; i++)
	{
		out[i] = in1[i] + in2[i];
	}
}


void cleanup()
{
	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{
		free(hostInput1);
		hostInput1 = NULL;
	}

}