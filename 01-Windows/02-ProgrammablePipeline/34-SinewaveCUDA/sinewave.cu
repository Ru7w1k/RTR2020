
#include <hip/hip_runtime.h>
// sinewave kernel
__global__ void sinewave(float4 *pos, unsigned int width, unsigned int height,
                         float animTime) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;

  u = (u * 2.0) - 1.0;
  v = (v * 2.0) - 1.0;

  float freq = 4.0f;
  float w = sinf(freq * u + animTime) * cosf(freq * v + animTime) * 0.5f;

  pos[y * width + x] = make_float4(u, w, v, 1.0f);
  return;
}

void launchCUDAKernel(float4 *pos, unsigned int width, unsigned int height,
                      float animTime) {
  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);

  sinewave<<<grid, block>>>(pos, width, height, animTime);
}
